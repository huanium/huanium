#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>  
#include <hipfft/hipfft.h>          // CUDA FFT Libraries
 
#include "defines.h"
#include "gp_kernels.h"
#include "chamber.hpp"


// *****************************
// Useful functions
//******************************
__device__
unsigned char clip(double x) {return x > 255 ? 255 : (x < 0 ? 0 : x); }

__device__
uchar4 viridis(double value) {
    uchar4 result;
    result.x = clip(255 * ( 2.854 * pow(value, 3) - 2.098 * pow(value, 2) + 0.037 * value + 0.254));
    result.y = clip(255 * (-0.176 * pow(value, 3) - 0.167 * pow(value, 2) + 1.243 * value + 0.016));
    result.z = clip(255 * ( 0.261 * pow(value, 3) - 1.833 * pow(value, 2) + 1.275 * value + 0.309));
    result.w = 255;
    return result;
}

__device__
uchar4 inferno(double value) {
    uchar4 result;
    result.x = clip(255 * (-1.760 * pow(value, 3) + 1.487  * pow(value, 2) + 1.223 * value - 0.034));
    result.y = clip(255 * ( 0.284 * pow(value, 3) - 0.827  * pow(value, 2) - 0.086 * value + 0.026));
    result.z = clip(255 * ( 7.533 * pow(value, 3) - 11.435 * pow(value, 2) + 4.603 * value - 0.096));
    result.w = 255;
    return result;
}


//Round a / b to nearest higher integer value
inline int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b)
{
    return (a % b != 0) ? (a - a % b + b) : a;
}

// Calculate the magnitude squared of a complex number
__host__ __device__
double complexMagnitudeSquared(hipDoubleComplex in){
	return in.x*in.x + in.y*in.y;
}



// *****************************
// GPU Kernels
//******************************

// Handles display mapping from hipDoubleComplex to uchar4
__global__
void display_psi(uchar4 *d_out, hipDoubleComplex *devPsi, double scale, int w, int h) { 
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing
    double mag = complexMagnitudeSquared(devPsi[i]);
    d_out[i] = viridis(mag/scale);
}

void colormapKernelLauncher(uchar4 *d_out, hipDoubleComplex *devPsi, double scale, int w, int h) {
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    display_psi<<<gridSize, blockSize>>>(d_out, devPsi, scale, w, h);
}


// Handles display mapping from hipDoubleComplex to RGBA int
__global__
void movie_frame(int *d_buffer, hipDoubleComplex *devPsi, double scale, int w, int h) { 
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing
    double mag = complexMagnitudeSquared(devPsi[i]);
    uchar4 d_out = viridis(mag/scale);
    d_buffer[i] = 65536 * d_out.z + 256 * d_out.y + d_out.x;
}

void movieFrameLauncher(int *d_buffer, hipDoubleComplex *devPsi, double scale, int w, int h) {
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    movie_frame<<<gridSize, blockSize>>>(d_buffer, devPsi, scale, w, h);
}


// Multiply the wavefunction with a real-valued scalar
__global__ 
void realmult_psi(hipDoubleComplex *devPsi, double mult, int w, int h) {
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing
    devPsi[i].x = devPsi[i].x * mult;
    devPsi[i].y = devPsi[i].y * mult;
}

void multKernelLauncher(hipDoubleComplex *devPsi, double mult, int w, int h){
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    realmult_psi<<<gridSize, blockSize>>>(devPsi, mult, w, h);
}



// Realspace evolution 
__global__
void realevolve_psi(hipDoubleComplex *devPsi, hipDoubleComplex *devExpPotential, hipDoubleComplex *out,
                    double g, double dt, double useReal, double cooling, int w, int h){
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing

    hipDoubleComplex tPotential = devExpPotential[i];
    hipDoubleComplex tPsi = devPsi[i];
    double gn = g * complexMagnitudeSquared(tPsi) * (dt / (2*HBAR));
    hipDoubleComplex expgn;
    expgn.x = exp( -gn * cooling) * cos( -gn * useReal);
    expgn.y = exp( -gn * cooling) * sin( -gn * useReal);

    hipDoubleComplex realspaceUnitary = hipCmul(tPotential, expgn);
    out[i] = hipCmul(realspaceUnitary, tPsi);
}

void realspaceKernelLauncher(hipDoubleComplex *devPsi, hipDoubleComplex *devExpPotential, hipDoubleComplex *out,
                             double g, double dt, double useReal, double cooling, int w, int h){
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    realevolve_psi<<<gridSize, blockSize>>>(devPsi, devExpPotential, out, g, dt, useReal, cooling, w, h);
}


// Momentum Space evolution 
__global__
void momentumevolve_psi(hipDoubleComplex *devPsi, hipDoubleComplex *devExpKinetic, hipDoubleComplex *out, int w, int h){
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing

    hipDoubleComplex tKinetic = devExpKinetic[i];
    hipDoubleComplex tPsi = devPsi[i];
    out[i] = hipCmul(tKinetic, tPsi);
}

void momentumspaceKernelLauncher(hipDoubleComplex *devPsi, hipDoubleComplex *devExpKinetic, hipDoubleComplex *out, int w, int h){
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    momentumevolve_psi<<<gridSize, blockSize>>>(devPsi, devExpKinetic, out, w, h);
}



// Realspace evolution 
__global__
void gaugefield(double omegaR, 
                double *devXkY, double *devYkX, hipDoubleComplex *devExpXkY, hipDoubleComplex *devExpYkX,
                double dt, double useReal, double cooling, int w, int h){
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing

    devExpXkY[i].x = exp( -omegaR * devXkY[i] * cooling * dt) * 
                     cos( -omegaR * devXkY[i] * useReal * dt);
    devExpXkY[i].y = exp( -omegaR * devXkY[i] * cooling * dt) * 
                     sin( -omegaR * devXkY[i] * useReal * dt);
    devExpYkX[i].x = exp( -omegaR * devYkX[i] * cooling * dt) * 
                     cos( -omegaR * devYkX[i] * useReal * dt);
    devExpYkX[i].y = exp( -omegaR * devYkX[i] * cooling * dt) * 
                     sin( -omegaR * devYkX[i] * useReal * dt);
}


void gaugefieldKernelLauncher(double omegaR, 
                              double *devXkY, double *devYkX, hipDoubleComplex *devExpXkY, hipDoubleComplex *devExpYkX,
                              double dt, double useReal, double cooling, int w, int h){
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    gaugefield<<<gridSize, blockSize>>>(omegaR, devXkY, devYkX, devExpXkY, devExpYkX, dt, useReal, cooling, w, h);
}



// Density psi
__global__ 
void density_psi(hipDoubleComplex *devPsi, double *density, int w, int h) {
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing

    density[i] = complexMagnitudeSquared(devPsi[i]);
}


//Normalization
__global__ 
void scalarDiv_wfcNorm(hipDoubleComplex *in, double dr, double* pSum, hipDoubleComplex *out, int w, int h){
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing

    hipDoubleComplex result;
    double norm = sqrt((pSum[0])*dr);
    result.x = (in[i].x/norm);
    result.y = (in[i].y/norm);
    out[i] = result;
}


/**
 * Routine for parallel summation. Can be looped over from host.
 * From GPUE-group (https://github.com/GPUE-group/GPUE)
 */
 __global__ void multipass(double* input, double* output){
    unsigned int tid = threadIdx.x + threadIdx.y*blockDim.x
                       + threadIdx.z * blockDim.x * blockDim.y;
    unsigned int bid = blockIdx.x + blockIdx.y * gridDim.x
                       + gridDim.x * gridDim.y * blockIdx.z;
    unsigned int gid = bid * (blockDim.x * blockDim.y * blockDim.z)
                    + (threadIdx.y * blockDim.x)
                    + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;

    extern __shared__ double sdatad[];
    sdatad[tid] = input[gid];
    __syncthreads();

    for(int i = blockDim.x>>1; i > 0; i>>=1){
        if(tid < i){
            sdatad[tid] += sdatad[tid + i];
        }
        __syncthreads();
    }
    if(tid==0){
        output[bid] = sdatad[0];
    }
}
/*
 * General-purpose summation of an array on the gpu, storing the result in the first element
*/
void gpuReduce(double* data, int length, int threadCount) {
    dim3 block(length / threadCount, 1, 1);
    dim3 threads(threadCount, 1, 1);

    while((double)length/threadCount > 1.0){
        multipass<<<block,threads,threadCount*sizeof(double)>>>(&data[0],
                                                                &data[0]);
        length /= threadCount;
        block = (int) ceil((double)length/threadCount);
    }
    multipass<<<1,length,threadCount*sizeof(double)>>>(&data[0],
                                                       &data[0]);
}

void parSum(hipDoubleComplex *devPsi, double *density, double dx, int w, int h){

    int DS = w * h;
    double dg = dx * dx;

    dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    dim3 grid_tmp(DS, 1, 1);
    dim3 threads(TILEX, TILEY);
    dim3 block(grid_tmp.x/threads.x, 1, 1);

    density_psi<<<gridSize, threads>>>(devPsi, density, w, h);

    gpuReduce(density, grid_tmp.x, threads.x);
/*
    // Writing out in the parSum Function (not recommended, for debugging)
    double *sum;
    sum = (double *) malloc(sizeof(double)*gsize);
    hipMemcpy(sum,density,sizeof(double)*gsize,
               hipMemcpyDeviceToHost);
    std::cout << (sum[0]) << '\n';
*/
    scalarDiv_wfcNorm<<<gridSize, threads>>>(devPsi, dg, density, devPsi, w, h);
}



// Spoon
__global__
void spoonKernel(double *devPotential, hipDoubleComplex *devExpPotential, spoonProps spoonP,
                 double dt, double useReal, double cooling, int w, int h){
    const int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((tidx >= w) || (tidy >= h)) return; // Check if in bounds
    const int i = tidx + tidy * w; // 1D indexing
    const int dist = (tidx - spoonP.pos.x) * (tidx - spoonP.pos.x) + (tidy - spoonP.pos.y) * (tidy - spoonP.pos.y);
    double spoon = spoonP.strength * exp( - pow(dist, 2) / pow(spoonP.radius, 2) );
    
    double devPotSpoon = devPotential[i];
    devPotSpoon += spoon;
    devExpPotential[i].x =  exp( -devPotSpoon * cooling * dt/(2*HBAR)) * 
                            cos( -devPotSpoon * useReal * dt/(2*HBAR));
    devExpPotential[i].y =  exp( -devPotSpoon * cooling * dt/(2*HBAR)) * 
                            sin( -devPotSpoon * useReal * dt/(2*HBAR));
}

void spoonKernelLauncher(double *devPotential, hipDoubleComplex *devExpPotential, spoonProps spoonP,
                        double dt, double useReal, double cooling, int w, int h) {
    const dim3 gridSize (iDivUp(w, TILEX), iDivUp(h, TILEY));
    const dim3 blockSize(TILEX, TILEY);
    spoonKernel<<<gridSize, blockSize>>>(devPotential, devExpPotential, spoonP, dt, useReal, cooling, w, h);
}



